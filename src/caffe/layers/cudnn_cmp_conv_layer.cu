
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_cmp_conv_layer.hpp"



namespace caffe {

__global__ void sync_cmp_conv_groups() { }

template <typename Dtype>
void CuDNNCmpConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
  int count = this->blobs_[0]->count();
  for (int i = 0; i < count; ++i)
  {
      muweight[i] *= this->masks_[i] ;
  }

  if(this->quantize_term_)
  {
    Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
    int count = this->blobs_[0]->count();

    for (int i = 0; i < count; ++i)
    {
      if (this->masks_[i])
        muweight[i] = this->centroids_[this->indices_[i]];
    }
  }
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_cmp_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNCmpConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  //LOG(INFO) << "CONV BACKWARD" ;
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  int count = 0 ;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
    count = this->blobs_[0]->count();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));

        Dtype* cpu_weight_diff = this->blobs_[0]->mutable_cpu_diff();

        for(int j = 0; j < count; ++j)
        {
          cpu_weight_diff[j] *= this->masks_[j];
        }


	if(this->quantize_term_)
	{
          vector<Dtype> tmpDiff(this->class_num_);
          vector<int> freq(this->class_num_);
          for (int j = 0; j < count; ++j)
          {
            if (this->masks_[j])
            {
               tmpDiff[this->indices_[j]] += cpu_weight_diff[j];
               freq[this->indices_[j]]++;
            }
          }
          for(int j = 0; j < count; ++j)
          {
            if (this->masks_[j])
            {
              cpu_weight_diff[j] = tmpDiff[this->indices_[j]]/freq[this->indices_[j]] ;
            }
          }
	}

      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_cmp_conv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNCmpConvolutionLayer);

}  // namespace caffe
#endif